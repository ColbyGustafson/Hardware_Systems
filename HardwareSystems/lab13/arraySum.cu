#include <stdio.h>
#include "hip/hip_runtime.h"

// Compile: nvcc -o arraySum arraySum.cu
// Run: ./arraySum

// Note: Host: CPU+MainMemory & Device: GPU

__global__ void addKernel(int *device_memory_a, int *device_memory_b, int *device_memory_c)
{   
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    device_memory_c[i] = device_memory_a[i] + device_memory_b[i];
}

// TODO: implement a kernel (like above) called squareKernel to square each element of an array
__global__ void squareKernel(int *device_memory_a)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	device_memory_a[i] = device_memory_a[i] * device_memory_a[i];
}


// function for using CUDA to add arrays in parallel
void addWithCuda(int *host_memory_a, int *host_memory_b, int *host_memory_c, size_t size)
{
	hipSetDevice(0);

    int *device_memory_a;
    int *device_memory_b;
    int *device_memory_c;
    
    
 	// Allocate memory in the GPU for two inputs and one output
    hipMalloc((void**)&device_memory_a, size * sizeof(int));
    hipMalloc((void**)&device_memory_b, size * sizeof(int));
    hipMalloc((void**)&device_memory_c, size * sizeof(int));


    // Copy input from CPU to GPU
    hipMemcpy(device_memory_a, host_memory_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_memory_b, host_memory_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    squareKernel<<<1, size>>>(device_memory_a);
    squareKernel<<<1, size>>>(device_memory_b);
    addKernel<<<1, size>>>(device_memory_a, device_memory_b, device_memory_c);
    hipDeviceSynchronize();

    // Copy output from GPU to CPU.
    hipMemcpy(host_memory_c, device_memory_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free memory in GPU
    hipFree(device_memory_a);
    hipFree(device_memory_b);
    hipFree(device_memory_c);
}

void fillArray(int *array, size_t size, int multiple)
{	
	int i;
	for(i=0; i<size; i++)
	{
		array[i] = (i+1)*multiple;
	}
}

void printArray(int *array, size_t size)
{
	int i;
	for(i=0; i<size; i++)
	{
		printf("%d, ", array[i]);
	}
	printf("\n");
}

int main()
{
    int arraySize = 50;
    int array_a[arraySize];
    int array_b[arraySize];
    int array_c[arraySize];

    fillArray(array_a, arraySize, 5);
    fillArray(array_b, arraySize, 7);

    printArray(array_a, arraySize); //should print multiples of 5
    printArray(array_b, arraySize); //should print multiples of 7

    // Add arrays in parallel
	addWithCuda(array_a, array_b, array_c, arraySize);

	printArray(array_c, arraySize);
    return 0;
}
