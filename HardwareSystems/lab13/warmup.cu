#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// Compile: nvcc -o warmup warmup.cu
// Run: ./warmup

__global__ void HelloCuda()
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    printf("Local thread Id = %d Block id = %d Global thread Id = ( %d )\n", threadIdx.x, blockIdx.x, i);
}


int main()
{

   int num_blocks = 6;
   int num_threads = 4;
   HelloCuda<<<num_blocks,num_threads>>>();
   hipDeviceSynchronize();
   
   printf("main done \n");

   return 0;
}
